#include "hip/hip_runtime.h"
#include <malloc.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"

#define MAX_BLKS 65537
#define MAX_THRDS_P_BLK 512

struct Vector
{
    float x;
    float y;
    float z;
    __device__ float d_influenceBy(Vector p)
    {
      return 1/sqrt((x-p.x)*(x-p.x)+(y-p.y)*(y-p.y)+(z-p.z)*(z-p.z));
    }
    __host__ float h_influenceBy(Vector p)
    {
      return 1/sqrt((x-p.x)*(x-p.x)+(y-p.y)*(y-p.y)+(z-p.z)*(z-p.z));
    }
};

__host__ int ciel(float value)
{
  float mantissa = value - (int)value;
  return ((int)value + (mantissa==0 ? 0 : 1));
}


/**
 * Kernel is launched with as many blocks as bodies. Not an optimal strategy but first iteration.
 * In each block, if number of threads is a power of two which is computed based on number of bodies given
 * Hence, the binary reduction in kernel is much simpler since thread load is power of two.
 * Each thread computes forces of <bodiesPerThread> bodies on itself, followed by a reduction to compute overall force.
 */
__global__ void forceComp(Vector *positions, int bodyCount, float* resultantForce, int bodiesPerThread)
{
  extern __shared__ float perBlockCache[];
  int tid = threadIdx.x*bodiesPerThread;
  int Limit = tid + bodiesPerThread;
  
  if( tid < bodyCount )
  {
    perBlockCache[threadIdx.x] = 0.0;
    while( tid < Limit )
    {
      if( blockIdx.x != tid )
        perBlockCache[threadIdx.x] += positions[blockIdx.x].d_influenceBy(positions[tid]);
      tid++;
    }
    __syncthreads();
  
    /* now do reduction by addition for the resultant
     * force on body with Id = blockIdx.x */
    int reduceDim = blockDim.x/2;
    while(reduceDim>0)
    {
      if( threadIdx.x < reduceDim )
		perBlockCache[threadIdx.x] += perBlockCache[threadIdx.x+reduceDim];
      __syncthreads();
      reduceDim /= 2;
    }
    if(threadIdx.x == 0)
      resultantForce[blockIdx.x] = perBlockCache[0];
  }
}

/**
 * Program requires a numerical input i.e. the number
 * of bodies participating in the n-body simulation
 * This number should be less than MAX_BLKS = 65537
 */
int main(int argc, char* argv[])
{
 if(argc == 2)
 {
  int host_bodyCount = atoi(argv[1]);
  if( host_bodyCount > MAX_BLKS )
  {
    printf("Please give a number N < %d\n", MAX_BLKS);
    return -1;
  }
  
  size_t 		res_size;
  int 			iter;
  float 		tempResult;
  Vector 		*host_positions;
  float 		*host_resultantForce;
  Vector 		*dev_positions;
  float 		*dev_resultantForce;
  hipEvent_t 	start, stop, startForceComp, stopForceComp;
  float 		total_time, timeForceComp;
  
  size_t 	size 			= host_bodyCount * sizeof(Vector);
  int 		blocksPerGrid 	= host_bodyCount;
  int thrdCntHold			= MAX_THRDS_P_BLK;
  for(int i=1; host_bodyCount<thrdCntHold && thrdCntHold>2 ; ++i) thrdCntHold >>= 1;
  int 		threadsPerBlock	= thrdCntHold;
  int 		bodiesPerThread = ciel((float)host_bodyCount/threadsPerBlock);
  res_size 					= threadsPerBlock*sizeof(float);

  printf("Blocks per Grid: %d\nThreads per Block: %d\n", blocksPerGrid, threadsPerBlock);
  srand(time(NULL));
  
  /* Allocate host memory to prepare data */
  host_positions = (Vector*)malloc(size);
  host_resultantForce = (float*)malloc(res_size);
  for( iter=0; iter < host_bodyCount; iter++ )
  { 
    host_positions[iter].x = iter+1.0;
    host_positions[iter].y = iter+1.0;
    host_positions[iter].z = iter+1.0;
    //printf("Body %d position is (%f,%f,%f)\n", iter+1, host_positions[iter].x, host_positions[iter].y, host_positions[iter].z );
  }
  
  /* Allocate device memory, GPU memory */
  hipMalloc((void**)&dev_positions, size);
  hipMalloc((void**)&dev_resultantForce, res_size);
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventCreate( &startForceComp );
  hipEventCreate( &stopForceComp );
  hipEventRecord( start, 0 );

  /* Copy data from host to device */
  hipMemcpy(dev_positions, host_positions, size, hipMemcpyHostToDevice );
   
  hipEventRecord( startForceComp, 0 );
  cudaPrintfInit();
  forceComp<<<blocksPerGrid, threadsPerBlock, res_size>>>(dev_positions, host_bodyCount, dev_resultantForce, bodiesPerThread);
  hipEventRecord( stopForceComp, 0 );
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();

   /* Copy result from device to host */
  hipMemcpy( host_resultantForce, dev_resultantForce, res_size, hipMemcpyDeviceToHost );

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &total_time, start, stop );
  hipEventSynchronize( stopForceComp );
  hipEventElapsedTime( &timeForceComp, startForceComp, stopForceComp );
  
  //for(iter = 0; iter < host_bodyCount; iter++)
   // printf("Force on me(%d) is %.3f\n", iter+1, host_resultantForce[iter]);
   
  printf("Time (data transfer+computation on device): %f ms\n", total_time);
  printf("Time (computation on device): %f ms\n", timeForceComp);
 
  /* Compute on host for comparison */
  int error = 0;
  hipEventRecord( startForceComp, 0 );
  for(int i=0; i< host_bodyCount; i++)
  {
    tempResult = 0.0;
    for (iter = 0; iter < host_bodyCount; iter++)
    { 
      if(iter != i)
        tempResult = tempResult + host_positions[i].h_influenceBy(host_positions[iter]);
    }
    //printf("Force on me(%d) is %.3f; Device result is %.3f\n", i+1, tempResult, host_resultantForce[i]);
  }
  printf("\n");
  if( error == 1 )
    printf("Noticeable error detected betweene host and devie computataion\n");
  hipEventRecord( stopForceComp, 0 );

  hipEventSynchronize( stopForceComp ); 
  hipEventElapsedTime( &timeForceComp, startForceComp, stopForceComp );
  printf("Time (computation done on host only): %f ms\n", timeForceComp);
  
  /* clear all memory  */
  hipFree(dev_positions);
  hipFree(dev_resultantForce);
  hipEventDestroy( start );
  hipEventDestroy( stop );
  hipEventDestroy( startForceComp );
  hipEventDestroy( stopForceComp );
  free(host_positions);
  free(host_resultantForce);
 }
 else
 {
   printf("Please provide atleast one argument.\n");
   return 1;
 }
 return 0;
}
